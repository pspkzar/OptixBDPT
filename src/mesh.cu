#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <hip/hip_math_constants.h>
#include "mesh.h"

#define BUMP_INTENSITY 0.1f

#define TESS 4

using namespace optix;

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, );

RT_PROGRAM void intersectMesh(int primIdx){
	//get indices
	int3 id=index_buffer[primIdx];
	//get vertices
	float3 v1=vertex_buffer[id.x];
	float3 v2=vertex_buffer[id.y];
	float3 v3=vertex_buffer[id.z];
	//intersect ray with triangle
	float3 n;
	float t, beta, gamma;
	if(intersect_triangle(current_ray, v1, v2, v3, n, t, beta, gamma))
	{
		//loading normals
		float3 n1=normal_buffer[id.x];
		float3 n2=normal_buffer[id.y];
		float3 n3=normal_buffer[id.z];

		float3 tmp_normal = (1.0f-beta-gamma)*n1 + beta*n2 +gamma*n3;


		//loading texCoords
		float2 tmp_texCoord;
		if(texCoord_buffer.size()==vertex_buffer.size()){
			float2 t1=texCoord_buffer[id.x];
			float2 t2=texCoord_buffer[id.y];
			float2 t3=texCoord_buffer[id.z];
			tmp_texCoord=(1.0f-beta-gamma)*t1 + beta*t2 +gamma*t3;
		}
		else
		{
			tmp_texCoord=make_float2(1.0f,0.0f);
		}

		float3 tmp_tangent, tmp_bitangent;

		if(tangent_buffer.size()==vertex_buffer.size()){
			float3 t1=tangent_buffer[id.x];
			float3 t2=tangent_buffer[id.y];
			float3 t3=tangent_buffer[id.z];

			float3 b1=bitangent_buffer[id.x];
			float3 b2=bitangent_buffer[id.y];
			float3 b3=bitangent_buffer[id.z];



			tmp_tangent=(1.0f-beta-gamma)*t1 + beta*t2 +gamma*t3;
			tmp_bitangent=(1.0f-beta-gamma)*b1 + beta*b2 +gamma*b3;


		}
		else{
			tmp_tangent=make_float3(0.f);
			tmp_bitangent=make_float3(0.f);
		}

		if(rtPotentialIntersection(t))
		{

			//setting attributes
			shading_normal=tmp_normal;
			geometric_normal=optix::normalize(n);
			texCoord=tmp_texCoord;
			tangent=tmp_tangent;
			bitangent=tmp_bitangent;
			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void boundingBoxMesh(int primIdx, float result[6]){
    //get indices
    int3 id=index_buffer[primIdx];
    //load vertices
    float3 v1=vertex_buffer[id.x];
    float3 v2=vertex_buffer[id.y];
    float3 v3=vertex_buffer[id.z];

    /*float3 v1d=vertex_buffer[id.x]-normal_buffer[id.x]* 1.5f * BUMP_INTENSITY;
    float3 v2d=vertex_buffer[id.y]-normal_buffer[id.y]* 1.5f * BUMP_INTENSITY;
    float3 v3d=vertex_buffer[id.z]-normal_buffer[id.z]* 1.5f * BUMP_INTENSITY;*/
    const float area = optix::length(optix::cross(v2-v1,v3-v1));
    Aabb* aabb = (optix::Aabb*)result;
    if(area>0.0f)
    {
        /*aabb->m_min=fminf(fminf(fminf(v1, v1d),fminf(v2, v2d)), fminf(v3, v3d));
        aabb->m_max=fmaxf(fmaxf(fmaxf(v1, v1d),fmaxf(v2, v2d)), fmaxf(v3, v3d));*/

        aabb->m_min=fminf(fminf(v1,v2), v3);
        aabb->m_max=fmaxf(fmaxf(v1,v2), v3);
    }
    else
    {
        aabb->invalidate();
    }
}
