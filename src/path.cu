#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "commonStructs.h"
#include "random.h"
#include "sphere_light.h"
#include "BSDF.h"

using namespace optix;

struct PathResult{
	float4 result;
	float4 atenuation;
	float3 position;
	float3 direction;
	unsigned int depth;
	unsigned int seed;
	float prob;
	bool count_emissive;
	bool finished;
};

struct ShadowResult{
	bool in_shadow;
};

#define MIN_DEPTH 1

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, );

//ray payloads
rtDeclareVariable(PathResult, current_path_result, rtPayload, );
rtDeclareVariable(ShadowResult, current_shadow_result, rtPayload, );

//kernel dimensions
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

//camera properties
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
//lens (for depth of field)
rtDeclareVariable(float, lens_radius, , )=0.0f;
rtDeclareVariable(float, focal_dist, , )=150.f;

//frame number to make sure result is different every frame
rtDeclareVariable(int, frame, , );
//samples for stratified sampling
rtDeclareVariable(int, sqrt_num_samples, , )=1;

//output buffer
rtBuffer<float4, 2> output;

//top object to start tracing rays
rtDeclareVariable(rtObject, top_object, , );


rtDeclareVariable(float, scene_epsilon, , )=0.01f;

RT_PROGRAM void camera(){
	unsigned int seed = tea<16>(launch_dim.x*launch_index.y+launch_index.x, frame);

	float2 inv_screen=1.f/(make_float2(launch_dim)) *2.f;
	float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;
	float2 jitter_scale = inv_screen / sqrt_num_samples;

	int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;

	float4 result = make_float4(0.0f);

	do{
		int x = samples_per_pixel%sqrt_num_samples;
		int y = samples_per_pixel/sqrt_num_samples;
		float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
		float2 d = pixel + jitter*jitter_scale;

		float r = lens_radius * sqrtf(rnd(seed));
		float ang = 2.f * M_PIf * rnd(seed);

		float3 ray_origin = eye + r * ( U * cosf(ang) + V * sinf(ang));
		float3 ray_target = eye + (d.x * U + d.y * V + W) * focal_dist;
		float3 ray_direction = normalize(ray_target - ray_origin);


		PathResult ray_result;
		ray_result.atenuation=make_float4(1.f);
		ray_result.count_emissive=true;
		ray_result.depth=0;
		ray_result.result=make_float4(0.f);
		ray_result.seed=seed;
		ray_result.finished=false;
		ray_result.prob=1.f;

		for(;;){

			Ray ray = make_Ray(ray_origin, ray_direction, PathRay, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, ray, ray_result);

			if(ray_result.finished)
				break;

			ray_origin=ray_result.position;
			ray_direction=ray_result.direction;
			ray_result.depth++;

		}

		result+=ray_result.result;
		seed=ray_result.seed;

	}while(--samples_per_pixel);

	result/=sqrt_num_samples * sqrt_num_samples;


	if(frame>1){
		float a = 1.f/float(frame);
		float b = float(frame-1)*a;
		float4 old_color=output[launch_index];///(make_float4(1.f) - output[launch_index]);
		float4 new_color = a*result+b*old_color;
		output[launch_index]= new_color;///(new_color + make_float4(1.f));
	}
	else{
		output[launch_index]=result;///(result + make_float4(1.f));
	}

}

RT_PROGRAM void exception(){
	//output[launch_index]=make_float4(1.f);
	rtPrintExceptionDetails();
}

RT_PROGRAM void path_miss(){
	current_path_result.finished = true;
}

#include "material.h"



RT_PROGRAM void glossy_shading(){
	//because we calculate direct lighting in every point of the path,
	//when first diffuse material is hit we stop counting emmisive contributions


	current_path_result.count_emissive=true;
	//calculate diffuse and specular probabilities.
	float4 diff_coef = Kd*tex2D(map_Kd, texCoord.x, texCoord.y);
	float4 spec_coef = Ks*tex2D(map_Ks, texCoord.x, texCoord.y);

	float3 position = current_ray.origin + current_ray.direction * t_hit;

	BSDF bsdf;
	bsdf.Kd=diff_coef;
	bsdf.Ks=spec_coef;
	bsdf.Ni=Ni;
	bsdf.Ns=Ns;

	float3 ffnormal = optix::faceforward(shading_normal, -current_ray.direction, shading_normal);

	for(int i=0; i<lights.size(); i++){
		//sample light
		float3 center = make_float3(lights[i].pos);

		float3 w = normalize(center-position);
		float3 v = normalize(cross(w, ffnormal));
		float3 u = cross(v, w);

		float u1, u2;
		u1=rnd(current_path_result.seed);
		u2=rnd(current_path_result.seed);

		float cos_a = 1 - u1 + u1 * sqrtf(1-powf(lights[i].pos.w / length(position-center), 2.f));
		float sin_a = sqrtf(1-cos_a*cos_a);
		float phi = 2 * M_PIf * u2;

		float3 dir = u * cosf(phi) * sin_a + v * sinf(phi) * sin_a + w * cos_a;
		float4 r;
		float pdf = bsdf.evaluate(current_ray.direction, shading_normal, dir, r);

		if(pdf>0.f){

			float radius = lights[i].pos.w;

			float3 o = position - center;

			float b = dot(o, dir);
			float c = dot(o, o) - radius * radius;
			float disc = b * b - c;

			float sdisc = sqrtf(disc);
			float root1 = (-b -sdisc);

			Ray shadow_test = make_Ray(position, dir, ShadowRay, scene_epsilon, root1);
			ShadowResult s_res;
			s_res.in_shadow=false;
			rtTrace(top_object, shadow_test, s_res);

			if(!s_res.in_shadow){

				float plight = radius / length(o);
				plight*=plight;
				plight=sqrtf(1.f-plight);
				plight=2.f*M_PIf * (1.f - plight);
				plight=1.f/plight;

				float weight = plight;
				weight/=weight + pdf;

				current_path_result.result += weight * (r * lights[i].color * current_path_result.atenuation * (1.f - sqrtf(1.f - powf(radius/length(position-center), 2.f))) * 2.f * M_PIf);
			}
		}
	}

	BSDF_Sample sample;
	sample.type=rnd(current_path_result.seed);
	sample.dir[0]=rnd(current_path_result.seed);
	sample.dir[1]=rnd(current_path_result.seed);

	float4 r;
	float3 out;

	float pdf=bsdf.sample(current_ray.direction, shading_normal, out, r, sample);

	if(pdf>0.f){
		current_path_result.atenuation*=r;
		current_path_result.direction=out;
		current_path_result.position=position;
		current_path_result.prob=pdf;
	}
	else{
		current_path_result.finished=true;
	}
}

RT_PROGRAM void path_ignore_alpha(){
	float4 color=Kd*tex2D(map_Kd, texCoord.x, texCoord.y);
	if(color.w == 0.f) rtIgnoreIntersection();
}


RT_PROGRAM void shadow_probe(){
	float4 color=Kd*tex2D(map_Kd, texCoord.x, texCoord.y);
	if(color.w == 0.f) rtIgnoreIntersection();
	else{
		current_shadow_result.in_shadow=true;
		rtTerminateRay();
	}
}

RT_PROGRAM void light_shading(){
	if(current_path_result.depth==0) {
		current_path_result.result += light.color * current_path_result.atenuation;
		current_path_result.finished=true;
		return;
	}

	float plight = light.pos.w / optix::length(current_ray.origin-make_float3(light.pos));
	plight*=plight;
	plight=sqrtf(1.f-plight);
	plight=2.f*M_PIf * (1.f - plight);
	plight=1.f/plight;

	float weight = current_path_result.prob;
	weight /= weight + plight;

	current_path_result.result += light.color * current_path_result.atenuation * weight;

	current_path_result.finished=true;
}

RT_PROGRAM void shadow_probe_light(){
	current_shadow_result.in_shadow=true;
	rtTerminateRay();
}


